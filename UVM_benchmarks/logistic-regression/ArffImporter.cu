#include "hip/hip_runtime.h"
#include "ArffImporter.cuh"
#include <hip/hip_runtime.h>

using namespace std;

ArffImporter::ArffImporter() {}

ArffImporter::~ArffImporter() {
  free(featureMat);
  hipFree(featureMatTrans);
  hipFree(classArr);

  for (char *classAttr : classVec)
    free(classAttr);
  classVec.clear();

  for (NumericAttr &feature : featureVec)
    free(feature.name);
  featureVec.clear();
}

void ArffImporter::BuildFeatureMatrix() {
  if (featureMat != nullptr || featureMatTrans != nullptr)
    return;

  // Include X0 to be multiplied with bias
  numFeatures++;
  featureMat = (float *)malloc(numInstances * numFeatures * sizeof(float));

  //   featureMatTrans = (float *)malloc(numInstances * numFeatures *
  //   sizeof(float));
  //   classArr = (unsigned short *)malloc(numInstances * sizeof(unsigned
  //   short));

  hipMallocManaged(&featureMatTrans,
                    numInstances * numFeatures * sizeof(float));
  hipMallocManaged(&classArr, numInstances * sizeof(unsigned short));

  for (unsigned int i = 0; i < numInstances; i++) {
    float *offset = featureMat + i * numFeatures;
    // Set X0 to 1
    offset[0] = 1.0f;
    memmove(offset + 1, instanceVec[i].featureAttrArray,
            (numFeatures - 1) * sizeof(float));
    classArr[i] = instanceVec[i].classIndex;
    free(instanceVec[i].featureAttrArray);
  }

  Normalize();
  Transpose();
  instanceVec.clear();
}

void ArffImporter::Normalize() {
  // Do not normalize X0
  for (unsigned int i = 1; i < numFeatures; i++) {
    // Use either range / standard deviation
    float range = featureVec[i - 1].max - featureVec[i - 1].min;
    if (range == 0.0)
      continue;

    for (unsigned int j = 0; j < numInstances; j++) {
      unsigned int featureIndex = j * numFeatures + i;
      featureMat[featureIndex] =
          (featureMat[featureIndex] - featureVec[i - 1].mean) / range;
    }
  }
}

void ArffImporter::Transpose() {
  for (unsigned int i = 0; i < numInstances; i++)
    for (unsigned int j = 0; j < numFeatures; j++)
      featureMatTrans[j * numInstances + i] = featureMat[i * numFeatures + j];
}

// Need to check string length boundary
void ArffImporter::Read(const char *fileName) {
  FILE *fp;

  if ((fp = fopen(fileName, "r+")) == nullptr) {
    printf("File: %s not found!\n", fileName);
    return;
  }

  // Assuming all data types of all features are float
  // and ignoring feature types
  char firstToken[TOKEN_LENGTH_MAX];
  char buffer[READ_LINE_MAX];

  while (fgets(buffer, READ_LINE_MAX, fp) != nullptr) {
    // Skip empty lines
    if (buffer[0] == '\n')
      continue;

    int readSize;
    sscanf(buffer, "%s%n", firstToken, &readSize);

    if (StrEqualCaseInsen(firstToken, KEYWORD_ATTRIBUTE)) {
      char *featureName = (char *)malloc(TOKEN_LENGTH_MAX);
      char *featureType = (char *)malloc(TOKEN_LENGTH_MAX);

      sscanf(buffer + readSize, "%s %s", featureName, featureType);

      // Read feature names
      if (StrEqualCaseInsen(featureType, KEYWORD_NUMERIC)) {
        // printf( "Feature name: %s, length: %d \n",
        //    featureName, GetStrLength( featureName ) );

        NumericAttr feature;
        feature.name = featureName;
        feature.min = 0.0;
        feature.max = 0.0;
        feature.mean = 0.0;
        featureVec.push_back(feature);
      }
      // Read class names
      else {
        // Parse classes attributes
        char *className = (char *)malloc(TOKEN_LENGTH_MAX);
        featureType++;

        while (sscanf(featureType, "%[^,}]%n", className, &readSize) > 0) {
          printf("Class name: %s \n", className);

          classVec.push_back(className);
          className = (char *)malloc(TOKEN_LENGTH_MAX);

          featureType += readSize + 1;
        }
      }

      continue;
    }
    // Read feature values
    else if (StrEqualCaseInsen(firstToken, KEYWORD_DATA)) {
      numFeatures = featureVec.size();
      numClasses = classVec.size();

      unsigned int featureAttrArraySize = numFeatures * sizeof(float);

      float *featureValueSumArr = (float *)calloc(numFeatures, sizeof(float));

      while (fgets(buffer, READ_LINE_MAX, fp) != nullptr) {
        unsigned int index = 0;
        unsigned int featureIndex = 0;
        float value;

        Instance instance;
        instance.featureAttrArray = (float *)malloc(featureAttrArraySize);

        // Get feature attribute value
        while (sscanf(buffer + index, "%f%n", &value, &readSize) > 0) {
          if (featureVec[featureIndex].min > value)
            featureVec[featureIndex].min = value;

          if (featureVec[featureIndex].max < value)
            featureVec[featureIndex].max = value;

          featureValueSumArr[featureIndex] += value;
          instance.featureAttrArray[featureIndex++] = value;
          index += readSize + 1;
        }

        // Get class attribute value
        char classValue[TOKEN_LENGTH_MAX];
        sscanf(buffer + index, "%s%n", classValue, &readSize);

        for (unsigned short i = 0; i < numClasses; i++) {
          if (StrEqualCaseSen(classVec[i], classValue)) {
            instance.classIndex = i;
            break;
          }
        }

        instanceVec.push_back(instance);
      }

      unsigned int instanceSize = instanceVec.size();

      // Compute bucket size and mean value for each numerical attribute
      for (unsigned int i = 0; i < numFeatures; i++) {
        featureVec[i].mean = featureValueSumArr[i] / instanceSize;

        // printf(
        //     "feature %u, max: %f, min: %f, mean: %f\n",
        //     i,
        //     featureVec[i].max,
        //     featureVec[i].min,
        //     featureVec[i].mean );
      }

      free(featureValueSumArr);
      featureValueSumArr = nullptr;

      break;
    }
  }

  numInstances = instanceVec.size();

  fclose(fp);
  BuildFeatureMatrix();
}

std::vector<char *> ArffImporter::GetClassAttr() { return classVec; }

std::vector<NumericAttr> ArffImporter::GetFeatures() { return featureVec; }

float *ArffImporter::GetFeatureMat() { return featureMat; }

float *ArffImporter::GetFeatureMatTrans() { return featureMatTrans; }

unsigned short *ArffImporter::GetClassIndex() { return classArr; }

unsigned int ArffImporter::GetNumInstances() { return numInstances; }

unsigned int ArffImporter::GetNumFeatures() { return numFeatures; }
