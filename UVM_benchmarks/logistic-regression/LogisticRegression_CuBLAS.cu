#include "hip/hip_runtime.h"
#include "ArffImporter.cuh"
#include "Helper.h"

#include <hipblas.h>

#define MAX_ITER 1000
#define LEARNING_RATE 10.0f

Node initNode(unsigned int numFeatures) {
  Node node;
  node.numFeatures = numFeatures;
  //   node.weights = (float *)malloc(numFeatures * sizeof(float));
  hipMallocManaged(&node.weights, numFeatures * sizeof(float));
  memset(node.weights, 0, numFeatures * sizeof(float));

  return node;
}

__global__ void ComputeCost(float *__restrict__ dCostArr,
                            const unsigned short *__restrict__ dClassArr,
                            const unsigned int numInstances) {
  unsigned int instanceId = blockIdx.x * blockDim.x + threadIdx.x;
  if (instanceId >= numInstances)
    return;

  float cost = dCostArr[instanceId];
  cost = 1.0f / (1.0f + expf(-cost)) - (float)dClassArr[instanceId];
  dCostArr[instanceId] = cost;
}

inline void cudaErrorCheck(hipError_t cudaStatus) {
  if (cudaStatus != hipSuccess)
    printf("kernel launch failed with error \"%s\".\n",
           hipGetErrorString(cudaStatus));
}

inline void cublasErrorCheck(hipblasStatus_t hipblasStatus_t) {
  if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) {
    printf("CuBLAS launch failed with error\n");
    switch (hipblasStatus_t) {
    case HIPBLAS_STATUS_NOT_INITIALIZED:
      printf("HIPBLAS_STATUS_NOT_INITIALIZED\n");

    case HIPBLAS_STATUS_ALLOC_FAILED:
      printf("HIPBLAS_STATUS_ALLOC_FAILED\n");

    case HIPBLAS_STATUS_INVALID_VALUE:
      printf("HIPBLAS_STATUS_INVALID_VALUE\n");

    case HIPBLAS_STATUS_ARCH_MISMATCH:
      printf("HIPBLAS_STATUS_ARCH_MISMATCH\n");

    case HIPBLAS_STATUS_MAPPING_ERROR:
      printf("HIPBLAS_STATUS_MAPPING_ERROR\n");

    case HIPBLAS_STATUS_EXECUTION_FAILED:
      printf("HIPBLAS_STATUS_EXECUTION_FAILED\n");

    case HIPBLAS_STATUS_INTERNAL_ERROR:
      printf("HIPBLAS_STATUS_INTERNAL_ERROR\n");
    }
  }
}

int main() {
  ArffImporter trainSetImporter;
  trainSetImporter.Read("Dataset/train/train-first1000.arff");

  // ArffImporter testSetImporter;
  // testSetImporter.Read( "Dataset/test/dev-first1000.arff" );

  // Init host data
  float *featureMatTrans = trainSetImporter.GetFeatureMatTrans();
  unsigned short *classArr = trainSetImporter.GetClassIndex();
  unsigned int numInstances = trainSetImporter.GetNumInstances();
  unsigned int numFeatures = trainSetImporter.GetNumFeatures();
  Node node = initNode(numFeatures);

  // Init device data
  float *dCostArr = nullptr;
  float *dWeightArr = node.weights;
  float *dFeaCostProdArr = nullptr;

  cudaErrorCheck(hipMallocManaged(&dCostArr, numInstances * sizeof(float)));

  cudaErrorCheck(
      hipMallocManaged(&dFeaCostProdArr, numFeatures * sizeof(float)));

  /* Determine block and grid size of ComputeCost kernel */
  dim3 ccBlockDim;
  dim3 ccGridDim;
  if (numInstances > 128) {
    ccBlockDim.x = 128;
    ccGridDim.x = (numInstances + 127) / 128;
  } else
    ccBlockDim.x = numInstances;

  // Init CuBLAS
  hipblasHandle_t cublasHandle;
  cublasErrorCheck(hipblasCreate(&cublasHandle));

  // Gradient descent params
  float updateWParam = -LEARNING_RATE / (float)numInstances;
  unsigned int iter = 0;

  time_t start, end;
  float dif;
  time(&start);

  printf("\nStart gradient descent...\n");

  float default_alpha = 1.0f;
  float default_beta = 0.0f;
  // Gradient descent
  while (iter++ < MAX_ITER) {
    // Classify
    cublasErrorCheck(hipblasSgemv(cublasHandle, HIPBLAS_OP_N, numInstances,
                                 numFeatures, &default_alpha, featureMatTrans,
                                 numInstances, dWeightArr, 1, &default_beta,
                                 dCostArr, 1));
    ComputeCost<<<ccGridDim, ccBlockDim>>>(dCostArr, classArr, numInstances);
    cudaErrorCheck(hipGetLastError());
    // Cost vec dot FeaMat-Transpose
    cublasErrorCheck(hipblasSgemv(cublasHandle, HIPBLAS_OP_T, numInstances,
                                 numFeatures, &default_alpha, featureMatTrans,
                                 numInstances, dCostArr, 1, &default_beta,
                                 dFeaCostProdArr, 1));
    // Update weights
    cublasErrorCheck(hipblasSaxpy(cublasHandle, numFeatures, &updateWParam,
                                 dFeaCostProdArr, 1, dWeightArr, 1));
  }
  cudaErrorCheck(hipDeviceSynchronize());

  cublasErrorCheck(hipblasDestroy(cublasHandle));
  // cudaErrorCheck(hipMemcpy(node.weights, dWeightArr,
  //                           numFeatures * sizeof(float),
  //                           hipMemcpyDeviceToHost));

  time(&end);
  dif = difftime(end, start);
  printf("Time taken is %.2lf seconds.\n", dif);

  printf("Updating weights completed, weight: %f\n", node.weights[0]);

  //   hipFree(dFeatureMatTrans);
  //   hipFree(dClassArr);
  //   hipFree(dWeightArr);
  hipFree(dCostArr);
  hipFree(dFeaCostProdArr);
  hipFree(node.weights);

  return 0;
}
